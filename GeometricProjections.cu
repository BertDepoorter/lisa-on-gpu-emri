#include "hip/hip_runtime.h"
#include "stdio.h"
#include "cuda_complex.hpp"
#include "GeometricProjections.hh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__ __device__
void get_basis_vecs(double lam, double beta, double u[], double v[], double k[])
{
	long i;

	double cosbeta, sinbeta, coslam, sinlam;

	for (i=0; i<3; i++)
	{
		u[i] = 0.;
		v[i] = 0.;
		k[i] = 0.;
	}

	cosbeta = cos(beta);
	sinbeta = sin(beta);

    coslam = cos(lam);
    sinlam = sin(lam);

	u[0] =  -sinbeta*coslam;  u[1] =  sinbeta*sinlam;  u[2] = cosbeta;
	v[0] =  sinlam;        v[1] = -coslam;        v[2] =  0.;
	k[0] = -cosbeta*coslam;  k[1] = -cosbeta*sinlam;  k[2] = -cosbeta;

	return;
}

__device__
double dot_product_1d(double *arr1, double *arr2){
    double out = 0.0;
    for (int i=0; i<3; i++){
        out += arr1[i]*arr2[i];
    }
    return out;
}


__device__
void xi_projections(double *xi_p, double *xi_c, double *u, double *v, double *n)
{
    double u_dot_n = dot_product_1d(u, n);
    double v_dot_n = dot_product_1d(v, n);

    *xi_p = (u_dot_n*u_dot_n) - (v_dot_n*v_dot_n);
    *xi_c = 2.0*u_dot_n*v_dot_n;
}

__device__
double interp_h(double delay, double out)
{

    return out;

}

<<<<<<< Updated upstream

=======
/*
>>>>>>> Stashed changes
__device__
void interp_single(double *result, double *input, int h, int d, double e, double *factorials, int start_input_ind)
{

	double A = 1.0;
	for (int i = 1; i < h; i += 1){
		A *= (i + e) * (i + 1 - e);
	}
	double denominator = factorials[h - 1] * factorials[h];
    A /= denominator;

	double B = 1.0 - e;
	double C = e;
	double D = e * (1.0 - e);

	double sum = 0.0;
    double temp_up, temp_down;
    //printf("in: %d %d\n", d, start_input_ind);
	for (int j = 1; j< h; j += 1){

		// get constants

		double first_term = factorials[h - 1] / factorials[h - 1 - j];
		double second_term = factorials[h] / factorials[h + j];
		double value = first_term * second_term;

		value = value * pow(-1.0, (double)j);

		double E = value;

		double F = j + e;
		double G = j + (1 - e);

        //printf("mid: %d %d %d\n", j, d, start_input_ind);

		// perform calculation
        temp_up = input[d + 1 + j - start_input_ind];
        temp_down = input[d - j - start_input_ind];
		sum += E * (temp_up / F + temp_down / G);

	}
    temp_up = input[d + 1 - start_input_ind];
    temp_down = input[d - start_input_ind];
    //printf("out: %d %d\n", d, start_input_ind);
	*result = A * (B * temp_up + C * temp_down + D * sum);
}
*/


__device__
void interp(double *result_hp, double *result_hc, cmplx *input, int h, int d, double e, double *factorials, int start_input_ind)
{

	double A = 1.0;
	for (int i = 1; i < h; i += 1){
		A *= (i + e) * (i + 1 - e);
	}
	double denominator = factorials[h - 1] * factorials[h];
    A /= denominator;

	double B = 1.0 - e;
	double C = e;
	double D = e * (1.0 - e);

	double sum_hp = 0.0;
    double sum_hc = 0.0;
    cmplx temp_up, temp_down;
    //printf("in: %d %d\n", d, start_input_ind);
	for (int j = 1; j< h; j += 1){

		// get constants

		double first_term = factorials[h - 1] / factorials[h - 1 - j];
		double second_term = factorials[h] / factorials[h + j];
		double value = first_term * second_term;

		value = value * pow(-1.0, (double)j);

		double E = value;

		double F = j + e;
		double G = j + (1 - e);

        //printf("mid: %d %d %d\n", j, d, start_input_ind);

		// perform calculation
        temp_up = input[d + 1 + j - start_input_ind];
        temp_down = input[d - j - start_input_ind];
		sum_hp += E * (temp_up.real() / F + temp_down.real() / G);
        sum_hc += E * (temp_up.imag() / F + temp_down.imag() / G);

	}
    temp_up = input[d + 1 - start_input_ind];
    temp_down = input[d - start_input_ind];
    //printf("out: %d %d\n", d, start_input_ind);
	*result_hp = A * (B * temp_up.real() + C * temp_down.real() + D * sum_hp);
    *result_hc = A * (B * temp_up.imag() + C * temp_down.imag() + D * sum_hc);
}

#define NUM_PARS  33
#define NUM_COEFFS 4
#define NLINKS  6


__global__
void response(double *y_gw, double *k_in, double *u_in, double *v_in, double dt,
              int num_delays, int *link_space_craft_0_in, int *link_space_craft_1_in,
              cmplx *input_in, int num_inputs, int order, double sampling_frequency,
              int buffer_integer, double *factorials_in, int num_factorials, double input_start_time,
              double* interp_array, double old_time, int old_ind, int start_ind, int end_ind, int init_length)
{


        __shared__ double factorials[100];
        __shared__ cmplx input[BUFFER_SIZE];
        __shared__ double first_delay;
        __shared__ double last_delay;
        __shared__ int start_input_ind;
        __shared__ int end_input_ind;
        __shared__ double spline_coeffs[NUM_PARS * NUM_COEFFS];


            __shared__ double k[3];
            __shared__ double u[3];
            __shared__ double v[3];
            __shared__ int link_space_craft_0[NLINKS];
            __shared__ int link_space_craft_1[NLINKS];

            __shared__ double x0_y[3];
            __shared__ double x0_c1[3];
            __shared__ double x0_c2[3];
            __shared__ double x0_c3[3];

            __shared__ double x1_y[3];
            __shared__ double x1_c1[3];
            __shared__ double x1_c2[3];
            __shared__ double x1_c3[3];

            __shared__ double n_y[3];
            __shared__ double n_c1[3];
            __shared__ double n_c2[3];
            __shared__ double n_c3[3];

            __shared__ double x0_all[3 * NUM_THREADS];
            __shared__ double x1_all[3 * NUM_THREADS];
            __shared__ double n_all[3 * NUM_THREADS];

            double* x0 = &x0_all[3 * threadIdx.x];
            double* x1 = &x1_all[3 * threadIdx.x];
            double* n = &n_all[3 * threadIdx.x];

            __shared__ double L_y, L_c1, L_c2, L_c3;

            double xi_p, xi_c;
            double k_dot_n, k_dot_x0, k_dot_x1;
            double t, L, delay0, delay1;
            double hp_del0, hp_del1, hc_del0, hc_del1;

            double large_factor, pre_factor;
            double clipped_delay0, clipped_delay1, out, fraction0, fraction1;
            int integer_delay0, integer_delay1, max_integer_delay, min_integer_delay;


        int start, end;

    __syncthreads();

    for (int i=threadIdx.x; i<3; i+=blockDim.x){
        k[i] = k_in[i];
        u[i] = u_in[i];
        v[i] = v_in[i];
         //if (threadIdx.x == 1) printf("%e %e %e\n", k[i], u[i], v[i]);
    }
    __syncthreads();

    for (int i=threadIdx.x; i<NLINKS; i+=blockDim.x){
        link_space_craft_0[i] = link_space_craft_0_in[i];
        link_space_craft_1[i] = link_space_craft_1_in[i];

        //if (threadIdx.x == 1) printf("%d %d %d %d\n", link_space_craft_0_in[i],link_space_craft_1_in[i], link_space_craft_1[i], link_space_craft_0[i]);
    }
    __syncthreads();

    for (int i = threadIdx.x; i<num_factorials; i += blockDim.x){
        factorials[i] = factorials_in[i];
    }
    __syncthreads();

    for (int i = threadIdx.x; i < NUM_PARS * NUM_COEFFS; i += blockDim.x)
      {
          int coeff_num = (int) (i / NUM_PARS);
          int par_num = i % NUM_PARS;

          int index = (coeff_num * init_length + old_ind) * NUM_PARS + par_num;

          spline_coeffs[par_num * 4 + coeff_num] = interp_array[index];

      }

    __syncthreads();


    int point_count = order + 1;
    int half_point_count = int(point_count / 2);

    for (int link_i=blockIdx.y; link_i<2; link_i+=gridDim.y){

        int sc0 = link_space_craft_0[link_i];
        int sc1 = link_space_craft_1[link_i];



        if (threadIdx.x == 0)
        {
            x0_y[0] = spline_coeffs[(3 * sc0 + 0) * 4 + 0];
            x0_y[1] = spline_coeffs[(3 * sc0 + 1) * 4 + 0];
            x0_y[2] = spline_coeffs[(3 * sc0 + 2) * 4 + 0];

            x0_c1[0] = spline_coeffs[(3 * sc0 + 0) * 4 + 1];
            x0_c1[1] = spline_coeffs[(3 * sc0 + 1) * 4 + 1];
            x0_c1[2] = spline_coeffs[(3 * sc0 + 2) * 4 + 1];

            x0_c2[0] = spline_coeffs[(3 * sc0 + 0) * 4 + 2];
            x0_c2[1] = spline_coeffs[(3 * sc0 + 1) * 4 + 2];
            x0_c2[2] = spline_coeffs[(3 * sc0 + 2) * 4 + 2];

            x0_c3[0] = spline_coeffs[(3 * sc0 + 0) * 4 + 3];
            x0_c3[1] = spline_coeffs[(3 * sc0 + 1) * 4 + 3];
            x0_c3[2] = spline_coeffs[(3 * sc0 + 2) * 4 + 3];

            x1_y[0] = spline_coeffs[(3 * sc1 + 0) * 4 + 0];
            x1_y[1] = spline_coeffs[(3 * sc1 + 1) * 4 + 0];
            x1_y[2] = spline_coeffs[(3 * sc1 + 2) * 4 + 0];

            x1_c1[0] = spline_coeffs[(3 * sc1 + 0) * 4 + 1];
            x1_c1[1] = spline_coeffs[(3 * sc1 + 1) * 4 + 1];
            x1_c1[2] = spline_coeffs[(3 * sc1 + 2) * 4 + 1];

            x1_c2[0] = spline_coeffs[(3 * sc1 + 0) * 4 + 2];
            x1_c2[1] = spline_coeffs[(3 * sc1 + 1) * 4 + 2];
            x1_c2[2] = spline_coeffs[(3 * sc1 + 2) * 4 + 2];

            x1_c3[0] = spline_coeffs[(3 * sc1 + 0) * 4 + 3];
            x1_c3[1] = spline_coeffs[(3 * sc1 + 1) * 4 + 3];
            x1_c3[2] = spline_coeffs[(3 * sc1 + 2) * 4 + 3];

            int start_ind = 3 * 3 * 4;
            n_y[0] = spline_coeffs[start_ind + (link_i * 3 + 0) * 4 + 0];
            n_y[1] = spline_coeffs[start_ind + (link_i * 3 + 1) * 4 + 0];
            n_y[2] = spline_coeffs[start_ind + (link_i * 3 + 2) * 4 + 0];

            n_c1[0] = spline_coeffs[start_ind + (link_i * 3 + 0) * 4 + 1];
            n_c1[1] = spline_coeffs[start_ind + (link_i * 3 + 1) * 4 + 1];
            n_c1[2] = spline_coeffs[start_ind + (link_i * 3 + 2) * 4 + 1];

            n_c2[0] = spline_coeffs[start_ind + (link_i * 3 + 0) * 4 + 2];
            n_c2[1] = spline_coeffs[start_ind + (link_i * 3 + 1) * 4 + 2];
            n_c2[2] = spline_coeffs[start_ind + (link_i * 3 + 2) * 4 + 2];

            n_c3[0] = spline_coeffs[start_ind + (link_i * 3 + 0) * 4 + 3];
            n_c3[1] = spline_coeffs[start_ind + (link_i * 3 + 1) * 4 + 3];
            n_c3[2] = spline_coeffs[start_ind + (link_i * 3 + 2) * 4 + 3];

            start_ind = start_ind + 6 * 3 * 4;

            L_y = spline_coeffs[start_ind + (link_i) * 4 + 0];
            L_c1 = spline_coeffs[start_ind + (link_i) * 4 + 1];
            L_c2 = spline_coeffs[start_ind + (link_i) * 4 + 2];
            L_c3 = spline_coeffs[start_ind + (link_i) * 4 + 3];

        }

        __syncthreads();

    for (int i=start_ind + threadIdx.x + blockDim.x*blockIdx.x;
         i < end_ind;
         i += blockDim.x * gridDim.x){

         int max_thread_num = (num_delays - blockDim.x*blockIdx.x > NUM_THREADS) ? NUM_THREADS : num_delays - blockDim.x*blockIdx.x;

         t = i*dt;

         // Interpolate everything
         double x_spl = t - old_time;
         double x2_spl = x_spl * x_spl;
         double x3_spl = x2_spl * x_spl;

         #pragma unroll
         for (int coord = 0; coord < 3; coord +=1)
         {
             x0[coord] = x0_y[coord] + x0_c1[coord] * x_spl + x0_c2[coord] * x2_spl + x0_c3[coord] * x3_spl;
             x1[coord] = x1_y[coord] + x1_c1[coord] * x_spl + x1_c2[coord] * x2_spl + x1_c3[coord] * x3_spl;
             n[coord] = n_y[coord] + n_c1[coord] * x_spl + n_c2[coord] * x2_spl + n_c3[coord] * x3_spl;
         }

         L = L_y + L_c1 * x_spl + L_c2 * x2_spl + L_c3 * x3_spl;

            //if (i <500) printf("%d %d: start \n", i, link_i);

         xi_projections(&xi_p, &xi_c, u, v, n);
         k_dot_n = dot_product_1d(k, n);
         k_dot_x0 = dot_product_1d(k, x0);
         k_dot_x1 = dot_product_1d(k, x1);

         delay0 = t - L - k_dot_x0*C_inv;
         delay1 = t - k_dot_x1*C_inv;


         clipped_delay0 = delay0 - input_start_time;
         integer_delay0 = (int) ceil(clipped_delay0 * sampling_frequency) - 1;
         fraction0 = 1.0 + integer_delay0 - clipped_delay0 * sampling_frequency;

         clipped_delay1 = delay1 - input_start_time;
         integer_delay1 = (int) ceil(clipped_delay1 * sampling_frequency) - 1;
         fraction1 = 1.0 + integer_delay1 - clipped_delay1 * sampling_frequency;

         max_integer_delay = (integer_delay0 < integer_delay1) ? integer_delay1 : integer_delay0;
         max_integer_delay += 2; // encompass all
         min_integer_delay = (integer_delay0 < integer_delay1) ? integer_delay0 : integer_delay1;

         if (threadIdx.x == 0){
              start_input_ind = min_integer_delay - buffer_integer;
        }

        if (threadIdx.x == max_thread_num - 1){
              end_input_ind = max_integer_delay + buffer_integer;
        }

        __syncthreads();

        //if (blockIdx.x == gridDim.x - 1) printf("%d %d %d %d %d %d %d %d %d %d\n", i, threadIdx.x, blockDim.x*blockIdx.x, num_delays, num_delays - blockDim.x*blockIdx.x, max_thread_num, start_input_ind, end_input_ind, integer_delay0, integer_delay1);
         for (int jj = threadIdx.x + start_input_ind; jj < end_input_ind; jj+=max_thread_num){
            //if (threadIdx.x == blockDim.x - 1) printf("%d, %d %d %d %d\n", blockIdx.x, link_i, jj - start_input_ind,  start_input_ind, end_input_ind);
            input[jj - start_input_ind] = input_in[jj];
         }


         __syncthreads();

         interp(&hp_del0, &hc_del0, input, half_point_count, integer_delay0, fraction0, factorials, start_input_ind);
         interp(&hp_del1, &hc_del1, input, half_point_count, integer_delay1, fraction1, factorials, start_input_ind);

         //hp_del0 = interp_h(delay0, 1.0);
         //if (i <500) printf("%d %d: %e \n", i, link_i, hp_del1);
         //hc_del0 = interp_h(delay0, 2.0);
         //hp_del1 = interp_h(delay1, 3.0);
         //hc_del1 = interp_h(delay1, 3.0);

         pre_factor = 1./(2*(1. - k_dot_n));
         large_factor = (hp_del0 - hp_del1)*xi_p + (hc_del0 - hc_del1)*xi_c;

         y_gw[link_i*num_delays + i] = pre_factor*large_factor;

         //printf("%d %e %e %e %e %e %e %e %e \n", threadIdx.x, pre_factor, hp_del0, hp_del1, hc_del0, hc_del1, xi_p, xi_c, large_factor);

         __syncthreads();
    }

}
        //double min_delay = (double) half_point_count / sampling_frequency;

}

// with uneven spacing in t in the sparse arrays, need to determine which timesteps the dense arrays fall into
// for interpolation
// effectively the boundaries and length of each interpolation segment of the dense array in the sparse array
void find_start_inds(int start_inds[], int unit_length[], double *t_arr, double delta_t, int *length, int new_length)
{

    double T = (new_length - 1) * delta_t;
  start_inds[0] = 0;
  int i = 1;
  for (i = 1;
       i < *length;
       i += 1){

          double t = t_arr[i];

          // adjust for waveforms that hit the end of the trajectory
          if (t < T){
              start_inds[i] = (int)std::ceil(t/delta_t);
              unit_length[i-1] = start_inds[i] - start_inds[i-1];
          } else {
            start_inds[i] = new_length;
            unit_length[i-1] = new_length - start_inds[i-1];
            break;
        }

      }

  // fixes for not using certain segments for the interpolation
  *length = i + 1;
}


void get_response(double *y_gw, double *k_in, double *u_in, double *v_in, double dt,
              int num_delays, int *link_space_craft_0_in, int *link_space_craft_1_in,
              cmplx *input_in, int num_inputs, int order,
              double sampling_frequency, int buffer_integer, double *factorials_in,
              int num_factorials, double input_start_time,
              double *interp_array, int init_len, double* h_t){

  int out_len = num_delays;

  // arrays for determining spline windows for new arrays
  int start_inds[init_len];
  int unit_length[init_len-1];

  int number_of_old_spline_points = init_len;

  // find the spline window information based on equally spaced new array
  find_start_inds(start_inds, unit_length, h_t, dt, &number_of_old_spline_points, out_len);

  #ifdef __HIPCC__

  // prepare streams for CUDA
  hipStream_t streams[number_of_old_spline_points-1];

  #endif

  #ifdef __USE_OMP__
  #pragma omp parallel for
  #endif
  for (int i = 0; i < number_of_old_spline_points-1; i++) {
        #ifdef __HIPCC__

        // create and execute with streams
        hipStreamCreate(&streams[i]);
        int num_blocks = std::ceil((unit_length[i] + NUM_THREADS -1)/NUM_THREADS);

        // sometimes a spline interval will have zero points
        if (num_blocks <= 0) continue;

        dim3 gridDim(num_blocks, NLINKS);

        //printf("RUNNING: %d\n", i);
        response<<<gridDim, NUM_THREADS>>>
                      (y_gw, k_in, u_in, v_in, dt,
                      num_delays, link_space_craft_0_in, link_space_craft_1_in,
                        input_in, num_inputs, order, sampling_frequency, buffer_integer,
                        factorials_in, num_factorials, input_start_time,
                        interp_array, h_t[i], i, start_inds[i], start_inds[i+1], init_len);
       #else

       // CPU waveform generation
       make_waveform(waveform,
                     interp_array,
                     M_phys, mu, qS, phiS, qK, phiK, dist,
                     nmodes, mich,
                     delta_t, h_t[i], i, start_inds[i], start_inds[i+1], init_len);
       #endif

    }

    //synchronize after all streams finish
    #ifdef __HIPCC__
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif
    for (int i = 0; i < number_of_old_spline_points-1; i++) {
          //destroy the streams
          hipStreamDestroy(streams[i]);
      }
    #endif
}


/*
int main()
{

	int num_fac = 100;
	double factorials_in[num_fac];

	factorials_in[0] = 1.0;

	for (int i=1; i<num_fac; i+=1){
		factorials_in[i] = i*factorials_in[i-1];
	}

	double *d_factorials_in;
	gpuErrchk(hipMalloc(&d_factorials_in, num_fac*sizeof(double)));
	gpuErrchk(hipMemcpy(d_factorials_in, factorials_in, num_fac*sizeof(double), hipMemcpyHostToDevice));

	int num_pts_in = int(1e6);


	cmplx *input_in = new cmplx[num_pts_in];
	cmplx *d_input_in;

	gpuErrchk(hipMalloc(&d_input_in, num_pts_in*sizeof(cmplx)));

	double sampling_frequency = 1.0;
	double dt = 1./sampling_frequency;
    double input_start_time = -10000.0;
    cmplx I(0.0, 1.0);
	for (int i=0; i<num_pts_in; i+=1) input_in[i] = sin(i*dt + input_start_time) + I*cos(i*dt + input_start_time);

	gpuErrchk(hipMemcpy(d_input_in, input_in, num_pts_in*sizeof(cmplx), hipMemcpyHostToDevice));

	int num_delays = int(1e5);

    int order = 25;
    int buffer_integer = order + 1;



    double beta = 0.5;
    double lam = 1.0;

    double k[3];
    double u[3];
    double v[3];

    get_basis_vecs(lam, beta, u, v, k);

    int nlinks = NLINKS;
    double *n_in = new double[num_delays*nlinks*3];
    double *x = new double[num_delays*3*3];
    double *L_vals = new double[num_delays*nlinks];
    int *link_space_craft_0 = new int[nlinks];
    int *link_space_craft_1 = new int[nlinks];

    link_space_craft_0[0] = 0; link_space_craft_1[0] = 1;
    link_space_craft_0[1] = 1; link_space_craft_1[1] = 0;

    link_space_craft_0[2] = 0; link_space_craft_1[2] = 2;
    link_space_craft_0[3] = 2; link_space_craft_1[3] = 0;

    link_space_craft_0[4] = 1; link_space_craft_1[4] = 2;
    link_space_craft_0[5] = 2; link_space_craft_1[5] = 1;

    double Re = 1.496e+11;  // meters
    double Phi0 = 0.0;

    double Omega0 = 1/(365.25*24.0*3600.0);

    double center_vec[3];

    double L = 2.5e9;

    double sc0_delta[2] = {L/2, -L/(2.*sqrt(3.))};

    double sc1_delta[2] = {-L/2, -L/(2.*sqrt(3.))};
    double sc2_delta[2] = {0.0, L/(sqrt(3.))};

    double Rnew, xnew, ynew, znew, t;
    double norm;
    int link_ind_0, link_ind_1;
    for (int i=0; i<num_delays; i++){
        t = i*dt;

        // sc 1
        Rnew = Re + sc0_delta[0];
        xnew = Rnew*cos(Omega0*t + Phi0);
        ynew = Rnew*sin(Omega0*t + Phi0);
        znew = sc0_delta[1];

        x[(0*3 + 0)*num_delays + i] = xnew;
        x[(0*3 + 1)*num_delays + i] = ynew;
        x[(0*3 + 2)*num_delays + i] = znew;

        Rnew = Re + sc1_delta[0];
        xnew = Rnew*cos(Omega0*t + Phi0);
        ynew = Rnew*sin(Omega0*t + Phi0);
        znew = sc1_delta[1];

        x[(1*3 + 0)*num_delays + i] = xnew;
        x[(1*3 + 1)*num_delays + i] = ynew;
        x[(1*3 + 2)*num_delays + i] = znew;

        Rnew = Re + sc2_delta[0];
        xnew = Rnew*cos(Omega0*t + Phi0);
        ynew = Rnew*sin(Omega0*t + Phi0);
        znew = sc2_delta[1];

        x[(2*3 + 0)*num_delays + i] = xnew;
        x[(2*3 + 1)*num_delays + i] = ynew;
        x[(2*3 + 2)*num_delays + i] = znew;

        for (int j=0; j<NLINKS; j++){
            link_ind_0 = link_space_craft_0[j];
            link_ind_1 = link_space_craft_1[j];

            xnew = x[(link_ind_0*3 + 0)*num_delays + i] - x[(link_ind_1*3 + 0)*num_delays + i];
            ynew = x[(link_ind_0*3 + 1)*num_delays + i] - x[(link_ind_1*3 + 1)*num_delays + i];
            znew = x[(link_ind_0*3 + 2)*num_delays + i] - x[(link_ind_1*3 + 2)*num_delays + i];

            norm = sqrt(xnew*xnew + ynew*ynew + znew*znew);

            n_in[(j*3 + 0)*num_delays + i] = xnew/norm;
            n_in[(j*3 + 1)*num_delays + i] = ynew/norm;
            n_in[(j*3 + 2)*num_delays + i] = znew/norm;
            L_vals[j*num_delays + i] = L;
        }
    }

    double *d_k, *d_u, *d_v, *d_x, *d_n_in;
    double *d_L_vals, *d_y_gw;

    int *d_link_space_craft_0, *d_link_space_craft_1;

    gpuErrchk(hipMalloc(&d_k, 3*sizeof(double)));
    gpuErrchk(hipMalloc(&d_u, 3*sizeof(double)));
    gpuErrchk(hipMalloc(&d_v, 3*sizeof(double)));

    gpuErrchk(hipMalloc(&d_x, 3*3*num_delays*sizeof(double)));
    gpuErrchk(hipMalloc(&d_n_in, nlinks*3*num_delays*sizeof(double)));

    gpuErrchk(hipMalloc(&d_link_space_craft_0, nlinks*sizeof(int)));
    gpuErrchk(hipMalloc(&d_link_space_craft_1, nlinks*sizeof(int)));

    gpuErrchk(hipMalloc(&d_L_vals, nlinks*num_delays*sizeof(double)));

    gpuErrchk(hipMemcpy(d_k, &k, 3*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, &u, 3*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, &v, 3*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_x, x, 3*3*num_delays*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_n_in, n_in, nlinks*3*num_delays*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_link_space_craft_0, link_space_craft_0, nlinks*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_link_space_craft_1, link_space_craft_1, nlinks*sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_L_vals, L_vals, num_delays*nlinks*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_y_gw, nlinks*num_delays*sizeof(double)));

    for (int i=0; i<1; i++){

        get_response(d_y_gw, d_k, d_u, d_v, dt, d_x, d_n_in,
                      num_delays, d_link_space_craft_0, d_link_space_craft_1,
                      d_L_vals, d_input_in, num_pts_in, order, sampling_frequency, buffer_integer, d_factorials_in, num_fac,  input_start_time);
}

    double *y_gw = new double[num_delays];

    gpuErrchk(hipMemcpy(y_gw, d_y_gw, num_delays*sizeof(double), hipMemcpyDeviceToHost));
    for (int i=0; i<1; i++) printf("%e\n", y_gw[i]);

    delete[] n_in;
    delete[] x;
    delete[] L_vals;
    delete[] link_space_craft_0;
    delete[] link_space_craft_1;

    gpuErrchk(hipFree(d_k));
    gpuErrchk(hipFree(d_u));
    gpuErrchk(hipFree(d_v));

    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_n_in));

    gpuErrchk(hipFree(d_link_space_craft_0));
    gpuErrchk(hipFree(d_link_space_craft_1));

    gpuErrchk(hipFree(d_L_vals));

    gpuErrchk(hipFree(d_y_gw));
    delete[] y_gw;

	gpuErrchk(hipFree(d_input_in));
	gpuErrchk(hipFree(d_factorials_in));

	delete[] input_in;
}
*/
