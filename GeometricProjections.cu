#include "hip/hip_runtime.h"
#include "stdio.h"
#include "cuda_complex.hpp"
#include "GeometricProjections.hh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__ __device__
void get_basis_vecs(double lam, double beta, double u[], double v[], double k[])
{
	long i;

	double cosbeta, sinbeta, coslam, sinlam;

	for (i=0; i<3; i++)
	{
		u[i] = 0.;
		v[i] = 0.;
		k[i] = 0.;
	}

	cosbeta = cos(beta);
	sinbeta = sin(beta);

    coslam = cos(lam);
    sinlam = sin(lam);

	u[0] =  -sinbeta*coslam;  u[1] =  sinbeta*sinlam;  u[2] = cosbeta;
	v[0] =  sinlam;        v[1] = -coslam;        v[2] =  0.;
	k[0] = -cosbeta*coslam;  k[1] = -cosbeta*sinlam;  k[2] = -cosbeta;

	return;
}

__device__
double dot_product_1d(double *arr1, double *arr2){
    double out = 0.0;
    for (int i=0; i<3; i++){
        out += arr1[i]*arr2[i];
    }
    return out;
}


__device__
void xi_projections(double *xi_p, double *xi_c, double *u, double *v, double *n)
{
    double u_dot_n = dot_product_1d(u, n);
    double v_dot_n = dot_product_1d(v, n);

    *xi_p = (u_dot_n*u_dot_n) - (v_dot_n*v_dot_n);
    *xi_c = 2.0*u_dot_n*v_dot_n;
}

__device__
double interp_h(double delay, double out)
{

    return out;

}

__device__
void interp(double *result_hp, double *result_hc, cmplx *input, int h, int d, double e, double *factorials, int start_input_ind)
{

	double A = 1.0;
	for (int i = 1; i < h; i += 1){
		A *= (i + e) * (i + 1 - e);
	}
	double denominator = factorials[h - 1] * factorials[h];
    A /= denominator;

	double B = 1.0 - e;
	double C = e;
	double D = e * (1.0 - e);

	double sum_hp = 0.0;
    double sum_hc = 0.0;
    cmplx temp_up, temp_down;
    //printf("in: %d %d\n", d, start_input_ind);
	for (int j = 1; j< h; j += 1){

		// get constants

		double first_term = factorials[h - 1] / factorials[h - 1 - j];
		double second_term = factorials[h] / factorials[h + j];
		double value = first_term * second_term;

		value = value * pow(-1.0, (double)j);

		double E = value;

		double F = j + e;
		double G = j + (1 - e);

        //printf("mid: %d %d %d\n", j, d, start_input_ind);

		// perform calculation
        temp_up = input[d + 1 + j - start_input_ind];
        temp_down = input[d - j - start_input_ind];
		sum_hp += E * (temp_up.real() / F + temp_down.real() / G);
        sum_hc += E * (temp_up.imag() / F + temp_down.imag() / G);

	}
    temp_up = input[d + 1 - start_input_ind];
    temp_down = input[d - start_input_ind];
    //printf("out: %d %d\n", d, start_input_ind);
	*result_hp = A * (B * temp_up.real() + C * temp_down.real() + D * sum_hp);
    *result_hc = A * (B * temp_up.imag() + C * temp_down.imag() + D * sum_hc);
}

__global__
void response(double *y_gw, double *k_in, double *u_in, double *v_in, double dt, double *x, double *n_in,
              int num_delays, int *link_space_craft_0_in, int *link_space_craft_1_in,
              double *L_vals, cmplx *input_in, int num_inputs, int order, double sampling_frequency, int buffer_integer, double *factorials_in, int num_factorials, double input_start_time)
{


        __shared__ double factorials[100];
        __shared__ cmplx input[1000];
        __shared__ double first_delay;
        __shared__ double last_delay;
        __shared__ int start_input_ind;
        __shared__ int end_input_ind;


            __shared__ double k[3];
            __shared__ double u[3];
            __shared__ double v[3];
            __shared__ double link_space_craft_0[6];
            __shared__ double link_space_craft_1[6];

            __shared__ double x0_all[3*NUM_THREADS];
            __shared__ double x1_all[3*NUM_THREADS];
            __shared__ double n_all[3*NUM_THREADS];

            double *x0 = &x0_all[3*threadIdx.x];
            double *x1 = &x1_all[3*threadIdx.x];
            double *n = &n_all[3*threadIdx.x];

            double xi_p, xi_c;
            double k_dot_n, k_dot_x0, k_dot_x1;
            double t, L, delay0, delay1;
            double hp_del0, hp_del1, hc_del0, hc_del1;

            double large_factor, pre_factor;
            double clipped_delay0, clipped_delay1, out, fraction0, fraction1;
            int integer_delay0, integer_delay1, max_integer_delay, min_integer_delay;


        int start, end;

    __syncthreads();

    for (int i=threadIdx.x; i<3; i+=blockDim.x){
        k[i] = k_in[i];
        u[i] = u_in[i];
        v[i] = v_in[i];
         //if (threadIdx.x == 1) printf("%e %e %e\n", k[i], u[i], v[i]);
    }
    __syncthreads();

    for (int i=threadIdx.x; i<6; i+=blockDim.x){
        link_space_craft_0[i] = link_space_craft_1_in[i];
        link_space_craft_1[i] = link_space_craft_0_in[i];
        //if (threadIdx.x == 1) printf("%d %d %d %d\n", link_space_craft_0_in[i],link_space_craft_1_in[i], link_space_craft_1[i], link_space_craft_0[i]);
    }
    __syncthreads();


    for (int i = threadIdx.x; i<num_factorials; i += blockDim.x){
        factorials[i] = factorials_in[i];
    }
    __syncthreads();

    int point_count = order + 1;
    int half_point_count = int(point_count / 2);

    for (int link_i=blockIdx.y; link_i<6; link_i+=gridDim.y){

        int sc0 = link_space_craft_0[link_i];
        int sc1 = link_space_craft_1[link_i];

    for (int i=threadIdx.x + blockDim.x*blockIdx.x;
         i < num_delays;
         i += blockDim.x*gridDim.x){

         int max_thread_num = (num_delays - blockDim.x*blockIdx.x > NUM_THREADS) ? NUM_THREADS : num_delays - blockDim.x*blockIdx.x;

         x0[0] = x[(sc0*3 + 0)*num_delays + i];
         x0[1] = x[(sc0*3 + 1)*num_delays + i];
         x0[2] = x[(sc0*3 + 2)*num_delays + i];

         x1[0] = x[(sc1*3 + 0)*num_delays + i];
         x1[1] = x[(sc1*3 + 1)*num_delays + i];
         x1[2] = x[(sc1*3 + 2)*num_delays + i];



         n[0] = n_in[(link_i*3 + 0)*num_delays + i];
         n[1] = n_in[(link_i*3 + 1)*num_delays + i];
         n[2] = n_in[(link_i*3 + 2)*num_delays + i];

         L = L_vals[link_i*num_delays + i];
         t = i*dt;
            //if (i <500) printf("%d %d: start \n", i, link_i);

         xi_projections(&xi_p, &xi_c, u, v, n);
         k_dot_n = dot_product_1d(k, n);
         k_dot_x0 = dot_product_1d(k, x0);
         k_dot_x1 = dot_product_1d(k, x1);

         delay0 = t - L - k_dot_x0*C_inv;
         delay1 = t - k_dot_x1*C_inv;

         clipped_delay0 = delay0 - input_start_time;
         integer_delay0 = (int) ceil(clipped_delay0 * sampling_frequency) - 1;
         fraction0 = 1.0 + integer_delay0 - clipped_delay0 * sampling_frequency;

         clipped_delay1 = delay1 - input_start_time;
         integer_delay1 = (int) ceil(clipped_delay1 * sampling_frequency) - 1;
         fraction1 = 1.0 + integer_delay1 - clipped_delay1 * sampling_frequency;

         max_integer_delay = (integer_delay0 < integer_delay1) ? integer_delay1 : integer_delay0;
         max_integer_delay += 2; // encompass all
         min_integer_delay = (integer_delay0 < integer_delay1) ? integer_delay0 : integer_delay1;

         if (threadIdx.x == 0){
              start_input_ind = min_integer_delay - buffer_integer;
        }
        if (threadIdx.x == max_thread_num - 1){
              end_input_ind = max_integer_delay + buffer_integer;
        }

        //printf("%d %d %d %d\n", integer_delay0, integer_delay1, start_input_ind, end_input_ind);

        __syncthreads();
        //if (blockIdx.x == gridDim.x - 1) printf("%d %d %d %d %d %d %d %d %d %d\n", i, threadIdx.x, blockDim.x*blockIdx.x, num_delays, num_delays - blockDim.x*blockIdx.x, max_thread_num, start_input_ind, end_input_ind, integer_delay0, integer_delay1);
         for (int jj = threadIdx.x + start_input_ind; jj < end_input_ind; jj+=max_thread_num){
            //if (threadIdx.x == blockDim.x - 1) printf("%d, %d %d %d %d\n", blockIdx.x, link_i, jj - start_input_ind,  start_input_ind, end_input_ind);
            input[jj - start_input_ind] = input_in[jj];
         }


         __syncthreads();

         interp(&hp_del0, &hc_del0, input, half_point_count, integer_delay0, fraction0, factorials, start_input_ind);
         interp(&hp_del1, &hc_del1, input, half_point_count, integer_delay1, fraction1, factorials, start_input_ind);

         //hp_del0 = interp_h(delay0, 1.0);
         //if (i <500) printf("%d %d: %e \n", i, link_i, hp_del1);
         //hc_del0 = interp_h(delay0, 2.0);
         //hp_del1 = interp_h(delay1, 3.0);
         //hc_del1 = interp_h(delay1, 3.0);

         pre_factor = 1./(2*(1. - k_dot_n));
         large_factor = (hp_del0 - hp_del1)*xi_p + (hc_del0 - hc_del1)*xi_c;

         y_gw[link_i*num_delays + i] = pre_factor*large_factor;

         //printf("%d %e %e %e %e %e %e %e %e \n", threadIdx.x, pre_factor, hp_del0, hp_del1, hc_del0, hc_del1, xi_p, xi_c, large_factor);


         __syncthreads();
    }
}


        //double min_delay = (double) half_point_count / sampling_frequency;

}


void get_response(double *y_gw, double *k_in, double *u_in, double *v_in, double dt, double *x, double *n_in,
              int num_delays, int *link_space_craft_0_in, int *link_space_craft_1_in,
              double *L_vals, cmplx *input_in, int num_inputs, int order, double sampling_frequency, int buffer_integer, double *factorials_in, int num_factorials, double input_start_time){

      int nblocks = (int) ceil((num_delays + NUM_THREADS - 1)/NUM_THREADS);

      dim3 gridDim(nblocks, 1);
      response<<<gridDim, NUM_THREADS>>>
      //response<<<1,1>>>
                    (y_gw, k_in, u_in, v_in, dt, x, n_in,
                    num_delays, link_space_craft_0_in, link_space_craft_1_in,
                    L_vals,
                      input_in, num_inputs, order, sampling_frequency, buffer_integer, factorials_in, num_factorials, input_start_time);

      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());

}


int main()
{

	int num_fac = 100;
	double factorials_in[num_fac];

	factorials_in[0] = 1.0;

	for (int i=1; i<num_fac; i+=1){
		factorials_in[i] = i*factorials_in[i-1];
	}

	double *d_factorials_in;
	gpuErrchk(hipMalloc(&d_factorials_in, num_fac*sizeof(double)));
	gpuErrchk(hipMemcpy(d_factorials_in, factorials_in, num_fac*sizeof(double), hipMemcpyHostToDevice));

	int num_pts_in = int(1e6);


	cmplx *input_in = new cmplx[num_pts_in];
	cmplx *d_input_in;

	gpuErrchk(hipMalloc(&d_input_in, num_pts_in*sizeof(cmplx)));

	double sampling_frequency = 1.0;
	double dt = 1./sampling_frequency;
    double input_start_time = -10000.0;
    cmplx I(0.0, 1.0);
	for (int i=0; i<num_pts_in; i+=1) input_in[i] = sin(i*dt + input_start_time) + I*cos(i*dt + input_start_time);

	gpuErrchk(hipMemcpy(d_input_in, input_in, num_pts_in*sizeof(cmplx), hipMemcpyHostToDevice));

	int num_delays = int(1e5);

    int order = 25;
    int buffer_integer = order + 1;



    double beta = 0.5;
    double lam = 1.0;

    double k[3];
    double u[3];
    double v[3];

    get_basis_vecs(lam, beta, u, v, k);

    int nlinks = 6;
    double *n_in = new double[num_delays*nlinks*3];
    double *x = new double[num_delays*3*3];
    double *L_vals = new double[num_delays*nlinks];
    int *link_space_craft_0 = new int[nlinks];
    int *link_space_craft_1 = new int[nlinks];

    link_space_craft_0[0] = 0; link_space_craft_1[0] = 1;
    link_space_craft_0[1] = 1; link_space_craft_1[1] = 0;

    link_space_craft_0[2] = 0; link_space_craft_1[2] = 2;
    link_space_craft_0[3] = 2; link_space_craft_1[3] = 0;

    link_space_craft_0[4] = 1; link_space_craft_1[4] = 2;
    link_space_craft_0[5] = 2; link_space_craft_1[5] = 1;

    double Re = 1.496e+11;  // meters
    double Phi0 = 0.0;

    double Omega0 = 1/(365.25*24.0*3600.0);

    double center_vec[3];

    double L = 2.5e9;

    double sc0_delta[2] = {L/2, -L/(2.*sqrt(3.))};

    double sc1_delta[2] = {-L/2, -L/(2.*sqrt(3.))};
    double sc2_delta[2] = {0.0, L/(sqrt(3.))};

    double Rnew, xnew, ynew, znew, t;
    double norm;
    int link_ind_0, link_ind_1;
    for (int i=0; i<num_delays; i++){
        t = i*dt;

        // sc 1
        Rnew = Re + sc0_delta[0];
        xnew = Rnew*cos(Omega0*t + Phi0);
        ynew = Rnew*sin(Omega0*t + Phi0);
        znew = sc0_delta[1];

        x[(0*3 + 0)*num_delays + i] = xnew;
        x[(0*3 + 1)*num_delays + i] = ynew;
        x[(0*3 + 2)*num_delays + i] = znew;

        Rnew = Re + sc1_delta[0];
        xnew = Rnew*cos(Omega0*t + Phi0);
        ynew = Rnew*sin(Omega0*t + Phi0);
        znew = sc1_delta[1];

        x[(1*3 + 0)*num_delays + i] = xnew;
        x[(1*3 + 1)*num_delays + i] = ynew;
        x[(1*3 + 2)*num_delays + i] = znew;

        Rnew = Re + sc2_delta[0];
        xnew = Rnew*cos(Omega0*t + Phi0);
        ynew = Rnew*sin(Omega0*t + Phi0);
        znew = sc2_delta[1];

        x[(2*3 + 0)*num_delays + i] = xnew;
        x[(2*3 + 1)*num_delays + i] = ynew;
        x[(2*3 + 2)*num_delays + i] = znew;

        for (int j=0; j<6; j++){
            link_ind_0 = link_space_craft_0[j];
            link_ind_1 = link_space_craft_1[j];

            xnew = x[(link_ind_0*3 + 0)*num_delays + i] - x[(link_ind_1*3 + 0)*num_delays + i];
            ynew = x[(link_ind_0*3 + 1)*num_delays + i] - x[(link_ind_1*3 + 1)*num_delays + i];
            znew = x[(link_ind_0*3 + 2)*num_delays + i] - x[(link_ind_1*3 + 2)*num_delays + i];

            norm = sqrt(xnew*xnew + ynew*ynew + znew*znew);

            n_in[(j*3 + 0)*num_delays + i] = xnew/norm;
            n_in[(j*3 + 1)*num_delays + i] = ynew/norm;
            n_in[(j*3 + 2)*num_delays + i] = znew/norm;
            L_vals[j*num_delays + i] = L;
        }
    }

    double *d_k, *d_u, *d_v, *d_x, *d_n_in;
    double *d_L_vals, *d_y_gw;

    int *d_link_space_craft_0, *d_link_space_craft_1;

    gpuErrchk(hipMalloc(&d_k, 3*sizeof(double)));
    gpuErrchk(hipMalloc(&d_u, 3*sizeof(double)));
    gpuErrchk(hipMalloc(&d_v, 3*sizeof(double)));

    gpuErrchk(hipMalloc(&d_x, 3*3*num_delays*sizeof(double)));
    gpuErrchk(hipMalloc(&d_n_in, nlinks*3*num_delays*sizeof(double)));

    gpuErrchk(hipMalloc(&d_link_space_craft_0, nlinks*sizeof(int)));
    gpuErrchk(hipMalloc(&d_link_space_craft_1, nlinks*sizeof(int)));

    gpuErrchk(hipMalloc(&d_L_vals, nlinks*num_delays*sizeof(double)));

    gpuErrchk(hipMemcpy(d_k, &k, 3*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, &u, 3*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, &v, 3*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_x, x, 3*3*num_delays*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_n_in, n_in, nlinks*3*num_delays*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_link_space_craft_0, link_space_craft_0, nlinks*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_link_space_craft_1, link_space_craft_1, nlinks*sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_L_vals, L_vals, num_delays*nlinks*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_y_gw, nlinks*num_delays*sizeof(double)));

    for (int i=0; i<1; i++){

        get_response(d_y_gw, d_k, d_u, d_v, dt, d_x, d_n_in,
                      num_delays, d_link_space_craft_0, d_link_space_craft_1,
                      d_L_vals, d_input_in, num_pts_in, order, sampling_frequency, buffer_integer, d_factorials_in, num_fac,  input_start_time);
}

    double *y_gw = new double[num_delays];

    gpuErrchk(hipMemcpy(y_gw, d_y_gw, num_delays*sizeof(double), hipMemcpyDeviceToHost));
    for (int i=0; i<1; i++) printf("%e\n", y_gw[i]);

    delete[] n_in;
    delete[] x;
    delete[] L_vals;
    delete[] link_space_craft_0;
    delete[] link_space_craft_1;

    gpuErrchk(hipFree(d_k));
    gpuErrchk(hipFree(d_u));
    gpuErrchk(hipFree(d_v));

    gpuErrchk(hipFree(d_x));
    gpuErrchk(hipFree(d_n_in));

    gpuErrchk(hipFree(d_link_space_craft_0));
    gpuErrchk(hipFree(d_link_space_craft_1));

    gpuErrchk(hipFree(d_L_vals));

    gpuErrchk(hipFree(d_y_gw));
    delete[] y_gw;

	gpuErrchk(hipFree(d_input_in));
	gpuErrchk(hipFree(d_factorials_in));

	delete[] input_in;
}
